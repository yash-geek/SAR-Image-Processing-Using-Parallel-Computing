#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/stat.h>
#include <errno.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "cJSON.h"

#ifdef _WIN32
#include <direct.h>
#define mkdir(dir, mode) _mkdir(dir)
#endif

#define PROGRESS_BAR_WIDTH 50
#define BLOCK_SIZE 16  // CUDA block size (16x16 threads)

// Error checking macro for CUDA calls
#define CUDA_CHECK(call) \
do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Function to print the progress bar
void printProgressBar(int current, int total) {
    float percentage = (float)current / total;
    int completed = (int)(percentage * PROGRESS_BAR_WIDTH);
    
    printf("\r[");  
    for (int i = 0; i < PROGRESS_BAR_WIDTH; i++) {
        if (i < completed) printf("=");
        else printf(" ");
    }
    printf("] %d%%", (int)(percentage * 100));
    fflush(stdout);
}

// Function to process the dataset and save the processed images in a different folder
void processDataset(const char *json_path, const char *image_dir, const char *output_dir) {
    clock_t start, end;
    double cpu_time_used;

    FILE *file = fopen(json_path, "r");
    if (!file) {
        printf("Could not open JSON file: %s\n", json_path);
        return;
    }

    fseek(file, 0, SEEK_END);
    long length = ftell(file);
    fseek(file, 0, SEEK_SET);
    char *json_data = (char *)malloc(length + 1);
    fread(json_data, 1, length, file);
    json_data[length] = '\0';
    fclose(file);

    cJSON *root = cJSON_Parse(json_data);
    if (!root) {
        printf("Error parsing JSON\n");
        free(json_data);
        return;
    }

    cJSON *images = cJSON_GetObjectItem(root, "images");
    if (!cJSON_IsArray(images)) {
        printf("Invalid JSON format\n");
        cJSON_Delete(root);
        free(json_data);
        return;
    }

    int total_images = cJSON_GetArraySize(images);
    int processed_images = 0;

    // Initialize CUDA
    CUDA_CHECK(hipSetDevice(0));

    start = clock(); // Start timing

    cJSON *image_item;
    cJSON_ArrayForEach(image_item, images) {
        cJSON *file_name = cJSON_GetObjectItem(image_item, "file_name");
        if (!cJSON_IsString(file_name)) continue;

        char image_path[512];
        char output_path[512];
        
        // Original image path
        sprintf(image_path, "%s/%s", image_dir, file_name->valuestring);
        
        // Output path for processed images
        sprintf(output_path, "%s/%s", output_dir, file_name->valuestring);

        int width, height, channels;
        unsigned char *image_data = stbi_load(image_path, &width, &height, &channels, STBI_grey);
        if (!image_data) {
            printf("\nCould not read image: %s\n", image_path);
            continue;
        }

        // Create the output directory if it doesn't exist
        #ifndef _WIN32
        mkdir(output_dir, 0777);
        #else
        mkdir(output_dir);
        #endif

        // Save processed image to the new location
        stbi_write_png(output_path, width, height, 1, image_data, width);
        stbi_image_free(image_data);

        // Update progress bar
        processed_images++;
        printProgressBar(processed_images, total_images);
    }

    cJSON_Delete(root);
    free(json_data);

    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    
    printf("\nProcessing time: %.3f seconds\n", cpu_time_used);
}

int main() {
    printf("Starting CUDA-accelerated model training...\n");

    processDataset("/content/dataset/SARscope/test/_annotations.coco.json",
    "/content/dataset/SARscope/test",
    "/content/dataset/processed_images");

    printf("\nTraining complete.\n");
    
    // Cleanup CUDA resources
    hipDeviceReset();
    
    return 0;
}
